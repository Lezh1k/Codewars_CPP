#include "hip/hip_runtime.h"
#include "rot13.cuh"
#include <assert.h>
#include <cstdint>

__device__ __forceinline__ char rot13_char(char c) {
  char cl = c | 0x20; // to lower
  int8_t is_alpha = (uint8_t)(cl - 'a') <= 'z' - 'a';
  int8_t offset = 13 - 26 * (cl > 'm');
  c += is_alpha * offset;
  return c;
}

__global__ void __cuda_rot13(char *str, size_t n) {
  // clang-format off
  // gridDim.x contains the size of the grid
  // blockIdx.x contains the index of the block with in the grid
  // blockDim.x contains the size of thread block (number of threads in the thread block).
  // threadIdx.x contains the index of the thread within the block
  // clang-format on
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n)
    return;
  str[idx] = rot13_char(str[idx]);
}
//////////////////////////////////////////////////////////////

__global__ void __cuda_rot13_vectorized(char *str, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t i = idx * 4;

  [[likely]]
  if (i + 3 < n) {
    uchar4 *vec = reinterpret_cast<uchar4 *>(str);
    uchar4 v = vec[idx];

    v.x = rot13_char(v.x);
    v.y = rot13_char(v.y);
    v.z = rot13_char(v.z);
    v.w = rot13_char(v.w);

    vec[idx] = v;
    return;
  }

  // Handle tail (non-multiple-of-4 end part)
  for (int j = 0; j < 4 && (i + j) < n; ++j) {
    str[i + j] = rot13_char(str[i + j]);
  }
}
//////////////////////////////////////////////////////////////

static const size_t gpu_buff_size = 1024ull * 1024ull * 1024ull * 2ull; // 2GB
void cuda_rot13_vect(char *str, size_t n) {
  size_t threads_n = 1024; // got from ktulhu
  size_t blocks_n = std::max(1ul, (n + 3) / 4 / threads_n);
  char *pd_str = nullptr;
  hipError_t err = hipMalloc((void **)&pd_str, gpu_buff_size);
  assert(err == hipSuccess);
  for (size_t i = 0; i < n; i += gpu_buff_size) {
    size_t to_copy = ((i + gpu_buff_size >= n) ? n - i : gpu_buff_size);
    err = hipMemcpy(pd_str, &str[i], sizeof(char) * to_copy,
                     hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    __cuda_rot13_vectorized<<<blocks_n, threads_n>>>(pd_str, to_copy);
    err = hipMemcpy(&str[i], pd_str, sizeof(char) * to_copy,
                     hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
  }
  hipFree((void *)pd_str);
}
//////////////////////////////////////////////////////////////

void cuda_rot13(char *str, size_t n) {
  size_t threads_n = 1024; // got from ktulhu
  size_t blocks_n = std::max(1ul, n / threads_n);
  char *pd_str = nullptr;
  hipError_t err = hipMalloc((void **)&pd_str, gpu_buff_size);

  assert(err == hipSuccess);
  for (size_t i = 0; i < n; i += gpu_buff_size) {
    size_t to_copy = ((i + gpu_buff_size >= n) ? n - i : gpu_buff_size);
    err = hipMemcpy(pd_str, &str[i], sizeof(char) * to_copy,
                     hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    __cuda_rot13<<<blocks_n, threads_n>>>(pd_str, to_copy);
    err = hipMemcpy(&str[i], pd_str, sizeof(char) * to_copy,
                     hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
  }
  hipFree((void *)pd_str);
}
//////////////////////////////////////////////////////////////
