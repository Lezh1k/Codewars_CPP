#include "hip/hip_runtime.h"
#include "rot13.h"
#include <assert.h>
#include <cstdint>

static const size_t threads_n = 1024;
__device__ __forceinline__ char rot13_char(char c) {
  char cl = c | 0x20; // to lower
  int8_t is_alpha = (uint8_t)(cl - 'a') <= 'z' - 'a';
  int8_t offset = 13 - 26 * (cl > 'm');
  c += is_alpha * offset;
  return c;
}

__global__ void __cuda_rot13(char *str, size_t n) {
  // clang-format off
  // gridDim.x contains the size of the grid
  // blockIdx.x contains the index of the block with in the grid
  // blockDim.x contains the size of thread block (number of threads in the thread block).
  // threadIdx.x contains the index of the thread within the block
  // clang-format on

  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n)
    return;
  str[idx] = rot13_char(str[idx]);
}
//////////////////////////////////////////////////////////////

__global__ void __cuda_rot13_vectorized(char *str, size_t n) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t n_vec = n / 4; // number of uchar4 elements
  uchar4 *__restrict__ vec = reinterpret_cast<uchar4 *>(str);

  // Vector path: each thread handles one uchar4
  if (tid < n_vec) {
    uchar4 v = vec[tid];
    v.x = rot13_char(v.x);
    v.y = rot13_char(v.y);
    v.z = rot13_char(v.z);
    v.w = rot13_char(v.w);
    vec[tid] = v;
  }

  // Tail (0–3 bytes) handled by a single thread to avoid races
  if (tid == 0) {
    const size_t base = n_vec * 4;
    for (size_t i = base; i < n; ++i) {
      str[i] = rot13_char(str[i]);
    }
  }
}
//////////////////////////////////////////////////////////////

static const size_t gpu_buff_max_size =
    1024ull * 1024ull * 1024ull * 2ull; // 2GB
void cuda_rot13_vect(char *str, size_t n) {
  char *pd_str;
  size_t n_vec = n / 4;
  size_t blocks_n = std::max(1ul, (n_vec + threads_n - 1) / threads_n);
  size_t gpu_buff_size = std::min(n, gpu_buff_max_size);
  hipError_t err = hipMalloc((void **)&pd_str, gpu_buff_size);
  assert(err == hipSuccess);

  for (size_t i = 0; i < n; i += gpu_buff_size) {
    size_t to_copy = ((i + gpu_buff_size >= n) ? n - i : gpu_buff_size);
    err = hipMemcpy(pd_str, &str[i], sizeof(char) * to_copy,
                     hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    __cuda_rot13_vectorized<<<blocks_n, threads_n>>>(pd_str, to_copy);
    err = hipMemcpy(&str[i], pd_str, sizeof(char) * to_copy,
                     hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
  }
  hipFree((void *)pd_str);
}
//////////////////////////////////////////////////////////////

void cuda_rot13(char *str, size_t n) {
  char *pd_str;
  size_t blocks_n = std::max(1ul, (n + threads_n) / threads_n);
  size_t gpu_buff_size = std::min(n, gpu_buff_max_size);
  hipError_t err = hipMalloc((void **)&pd_str, gpu_buff_size);
  assert(err == hipSuccess);

  for (size_t i = 0; i < n; i += gpu_buff_size) {
    size_t to_copy = ((i + gpu_buff_size >= n) ? n - i : gpu_buff_size);
    err = hipMemcpy(pd_str, &str[i], sizeof(char) * to_copy,
                     hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    __cuda_rot13<<<blocks_n, threads_n>>>(pd_str, to_copy);
    err = hipMemcpy(&str[i], pd_str, sizeof(char) * to_copy,
                     hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
  }
  hipFree((void *)pd_str);
}
//////////////////////////////////////////////////////////////
